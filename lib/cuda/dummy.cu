#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void 
vector_add_free_function_gpu(
    const int numel,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> A, // 1 is dimension of tensor
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> B,
    //output
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> out
    ) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x; //each thread will deal with a new value

    if(idx>=numel){ //don't go out of bounds
        return;
    }
    out[idx] = A[idx] +  B[idx];
}


using torch::Tensor;


template <typename T>
T div_round_up(T val, T divisor) {
	return (val + divisor - 1) / divisor;
}

torch::Tensor vector_add_free_function(const torch::Tensor A, const torch::Tensor B){
    CHECK(A.is_cuda()) << "A should be in GPU memory! Please call .cuda() on the tensor.";
    CHECK(B.is_cuda()) << "B should be in GPU memory! Please call .cuda() on the tensor.";
    
    torch::Tensor out = torch::empty_like(A);
    CHECK(out.is_cuda()) << "out should be in GPU memory! Please call .cuda() on the tensor.";
    int numel = A.size(0);

    const dim3 blocks = {(unsigned int)div_round_up(numel, BLOCK_SIZE), 1, 1};

    vector_add_free_function_gpu<<<blocks, BLOCK_SIZE>>>(
        numel,
        A.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
        B.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
        out.packed_accessor32<float,1,torch::RestrictPtrTraits>()
    );
    return out;
}
